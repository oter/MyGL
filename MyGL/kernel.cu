#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "kernels_common.cuh"


__global__ void clearColorKernel(uchar4* frameBuffer, const uchar4 color, int maxIndex)
{
	int offset = blockDim.x * blockIdx.x + threadIdx.x;

	if (offset >= maxIndex)
	{
		return;
	}
	

	frameBuffer[offset] = color;
}

__global__ void drawLinesKernel(uchar4* frameBuffer, int width, const int* lines, const uchar4* colors, int count)
{
	int offset = blockDim.x * blockIdx.x + threadIdx.x;

	

	

	if (offset >= count)
	{
		return;
	}

	

	int x0 = lines[offset * 4];
	int y0 = lines[offset * 4 + 1];
	int x1 = lines[offset * 4 + 2];
	int y1 = lines[offset * 4 + 3];

	


	//int sampleColor = 0xFFCC1923;
	//frameBuffer[x0 + y0 * width] = *(uchar4*)&sampleColor;
	//frameBuffer[x1 + y1 * width] = *(uchar4*)&sampleColor;

	uchar4 color = colors[offset];

	

	char steep = abs(y1 - y0) > abs(x1 - x0) ? 1 : 0; // ��������� ���� ������� �� ��� ��� � �� ��� �����
	// �������� ����� �� ���������, ���� ���� ������� ������� �������
	if (steep)
	{
		cudaSwap(x0, y0); // ����������� ��������� �������� � ��������� ������� ��� �������
		cudaSwap(x1, y1);
	}
	// ���� ����� ����� �� ����� �������, �� ������ ������ � ����� ������� �������
	if (x0 > x1)
	{
		cudaSwap(x0, x1);
		cudaSwap(y0, y1);
	}
	int dx = x1 - x0;
	int dy = abs(y1 - y0);
	int error = dx / 2; // ����� ������������ ����������� � ���������� �� dx, ����� ���������� �� ������ ������
	int ystep = (y0 < y1) ? 1 : -1; // �������� ����������� ����� ���������� y
	int y = y0;

	//for (int i = 0; i < 50; i++)
	//{
	//	frameBuffer[offset * width * 2 + i + 10] = color;
	//}

	//frameBuffer[offset] = { 0xFF, 0, 0, 0xFF };

	//return;

	for (int x = x0; x <= x1; x++)
	{
		int framebufferOffset = (steep ? y : x) + width * (steep ? x : y);
		frameBuffer[framebufferOffset] = color;
		error -= dy;
		if (error < 0)
		{
			y += ystep;
			error += dx;
		}
	}

	
}

void drawLinesKernel(uchar4* frameBuffer, dim3 threads, dim3 blocks, int width, const int* lines, const uchar4* colors, int count)
{
	drawLinesKernel << <blocks, threads >> >(frameBuffer, width, lines, colors, count);
}

void clearColorKernel(uchar4* frameBuffer, dim3 threads, dim3 blocks, const uchar4 color, int maxIndex)
{
	clearColorKernel << <blocks, threads >> >(frameBuffer, color, maxIndex);
}


//	TODO
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }