#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void clearColorKernel(uchar4* frameBuffer, const uchar4 color, int maxIndex)
{
	int offset = blockDim.x * blockIdx.x + threadIdx.x;

	if (offset >= maxIndex)
	{
		return;
	}

	frameBuffer[offset] = color;
}

void clearColorKernel(uchar4* frameBuffer, dim3 threads, dim3 blocks, const uchar4 color, int maxIndex)
{
	clearColorKernel << <blocks, threads >> >(frameBuffer, color, maxIndex);
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
