#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "kernels_common.cuh"
#include "TrianglesRaster.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math_functions.h"
#include ""

__global__ void trianglesRaster2DKernel(uchar4* frameBuffer, float2* vertices, int* indices, float2* normals, int2* textCoords, int count, int width, int height)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	

	if (i >= count)
	{
		return;
	}
	frameBuffer[i] = { 0, 0, 255, 0 };
	int2 p1, p2, p3;
	const int index = i * 3;
	p1.x = ((vertices[indices[index]].x + 1.0f) / 2) * width;
	p1.y = ((vertices[indices[index]].y + 1.0f) / 2) * height;

	p2.x = ((vertices[indices[index + 1]].x + 1.0f) / 2) * width;
	p2.y = ((vertices[indices[index + 1]].y + 1.0f) / 2) * height;

	p3.x = ((vertices[indices[index + 2]].x + 1.0f) / 2) * width;
	p3.y = ((vertices[indices[index + 2]].y + 1.0f) / 2) * height;

	if ((p1.y == p2.y) && (p1.y == p3.y))
	{
		return; // Invalid triangle
	}

	// sort the vertices, (x1, y1), (x2, y2), (x3, y3)
	if (p1.y > p2.y)
	{
		cudaSwap(p1, p2);
	}
	if (p1.y > p3.y)
	{
		cudaSwap(p1, p3);
	}
	if (p2.y > p3.y)
	{
		cudaSwap(p2, p3);
	}

	int total_height = p3.y - p1.y;
	for (int i = 0; i < total_height; i++) 
	{
		bool second_half = i > p2.y - p1.y || p2.y == p1.y;
		int segment_height = second_half ? p3.y - p2.y : p2.y - p1.y;
		float alpha = (float)i / total_height;
		float beta = (float)(i - (second_half ? p2.y - p1.y : 0)) / segment_height; // be careful: with above conditions no division by zero here

		//int ax1 = x1 + ()
		int2 a = cudaAdd(p1, cudaMult(cudaSub(p3, p1), alpha));
		int2 b = second_half ? cudaAdd(p2, cudaMult(cudaSub(p3, p2), beta)) : cudaAdd(p1, cudaMult(cudaSub(p2, p1), beta));
		if (a.x > b.x)
		{
			cudaSwap(a, b);
		}
		for (int j = a.x; j <= b.x; j++) 
		{
			int offset = (p1.y + i) * width + j;
			if (offset >= width * height)
			{
				continue;
			}
			frameBuffer[offset] = {255, 0, 0, 0};
			//image.set(j, t0.y + i, color); // attention, due to int casts t0.y+i != A.y
		}
	}



	//int a = 
}

void trianglesRaster2DKernel(std::vector<CUDA_ObjectBuffer>& objects, dim3 blocks, dim3 threads, uchar4* frameBuffer, int width, int height)
{
	for (size_t i = 0; i < objects.size(); i++)
	{
		
		trianglesRaster2DKernel << <blocks, threads >> >(frameBuffer, (float2*)objects.at(i).vertices.getMemPointer(), (int*)objects.at(i).indices.getMemPointer(),
			(float2*)objects.at(i).normals.getMemPointer(), (int2*)objects.at(i).textCoords.getMemPointer(), objects.at(i).count, width, height);
	}


}